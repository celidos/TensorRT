#include "hip/hip_runtime.h"
#include "kernel.h"

template <unsigned nthdsPerCTA>
__launch_bounds__(nthdsPerCTA) __global__
    void SeLUKernel(const int n, const float* input, float* output)
{
    for (int i = blockIdx.x * nthdsPerCTA + threadIdx.x; i < n; i += gridDim.x * nthdsPerCTA)
    {
        float mx = input[i] > 0.0f ? input[i] : 0.0f;
	float mn = 1.673263242f * (exp(input[i]) - 1.0f);
	if (mn < 0.0f) {
	    mn = 0.0f;
	}
	output[i] = 1.05070098f * (mx + mn);
    }
}

pluginStatus_t SeLUGPU(hipStream_t stream, const int n, const void* input, void* output)
{
    const int BS = 512;
    const int GS = (n + BS - 1) / BS;
    SeLUKernel<BS><<<GS, BS, 0, stream>>>(n,
                                           (const float*) input,
                                           (float*) output);
    return STATUS_SUCCESS;
}

pluginStatus_t SeLUInference(
    hipStream_t stream, const int n, const void* input, void* output)
{
    return SeLUGPU(stream, n, (const float*) input, (float*) output);
}
